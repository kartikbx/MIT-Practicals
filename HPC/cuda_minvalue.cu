    #include <hip/hip_runtime.h>
    #include <stdio.h>
    #include <time.h>

    #define SIZE 100

    __global__ void max(int *a , int *c)	// kernel function definition
    {
    int i = threadIdx.x;					// initialize i to thread ID

    *c = a[55];

            if(a[i] < *c)
                    {
                    *c = a[i];
                    }

    }

    int main()
    {
    int i;
    srand(time(NULL));		//makes use of the computer's internal clock to control the choice of the seed

    int a[SIZE];
    int c;

    int *dev_a, *dev_c;		//GPU / device parameters

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));		//assign memory to parameters on GPU from CUDA runtime API
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

    

    for( i = 0 ; i < SIZE ; i++)
    {
    	a[i] = rand();			// input the numbers
    }
    for( i = 0 ; i < SIZE ; i++)
    {
    	printf("%d", a[i]);			// input the numbers
    }
    
    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);		//copy the array from CPU to GPU
    max<<<1,SIZE>>>(dev_a,dev_c);										// call kernel function <<<number of blocks, number of threads
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

    printf("\nmin =  %d ",c);

    hipFree(dev_a);		// Free the allocated memory
    hipFree(dev_c);
    printf("");

    return 0;
    }
